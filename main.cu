#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include "matrix_build.h"
#include <hipblas.h>
#include <magma_lapack.h>
#include <magma_v2.h>
using namespace std;



int main() {
    
    

    magma_init();
    
    magma_int_t dev_t = 0;
    magma_queue_t queue_qr = NULL;
    magma_queue_create(dev_t,&queue_qr);
   
   //set up device and queue

    
    
    int numFluid = 2268;
    int numBoundary = 1903;
    int numGhost = 0;

    int numParticle = numFluid + numBoundary + numGhost;
    
    int numNeighbourone = 177;
    int numNeighbour = 177;
// read data from txt file
    double* inPressure = new double[numParticle];
    double* inVolume = new double[numParticle];
    double* inSoundSpeed = new double[numParticle];
    double* inVelocity = new double[numParticle];
    int* neighbourlist0 = new int[numFluid*numNeighbourone];
    int* neighbourlist1 = new int[numFluid*numNeighbourone];
    int* neighboursize0 =new int[numFluid];
    int* neighboursize1 = new int[numFluid];
    int* LPFOrder0 = new int[numFluid];
    int* LPFOrder1 = new int[numFluid];
    double* xPosition = new double[numParticle];
    double* yPosition = new double[numParticle];
    
    
    //store data into array

    ifstream myfile;
    
    myfile.open("xPosition.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        xPosition[i] = tem;
    }
    myfile.close();

    myfile.open("yPosition.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        yPosition[i] = tem;
    }
    myfile.close();


   myfile.open("inPressure.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        inPressure[i]=tem;
    }
    myfile.close();
    
    myfile.open("inVelocity.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        inVelocity[i]=tem;
    }
    myfile.close();
    
    myfile.open("inSoundSpeed.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        inSoundSpeed[i]=tem;
    }
    myfile.close();
   
    myfile.open("inVolume.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        inVolume[i]=tem;
    }
    myfile.close();
   
  myfile.open("neighbourlist0.txt");
    for(int i=0;i<numFluid*numNeighbourone;i++){
        int tem;
        myfile>>tem;
        neighbourlist0[i]=tem;
    }
    myfile.close();
    myfile.open("neighbourlist1.txt");
    for(int i=0;i<numFluid*numNeighbourone;i++){
        int tem;
        myfile>>tem;
        neighbourlist1[i]=tem;
    }
    myfile.close();
      myfile.open("neighboursize0.txt");
    for(int i=0;i<numFluid;i++){
       double tem;
        myfile>>tem;
        neighboursize0[i]=tem;
    }
    myfile.close();
     myfile.open("neighboursize1.txt");
    for(int i=0;i<numFluid;i++){
       double tem;
        myfile>>tem;
        neighboursize1[i]=tem;
    }
    myfile.close();


    fill_n(LPFOrder0,numFluid,1);
    fill_n(LPFOrder1,numFluid,1);

//device arrays which need copy
    
    double* d_xPosition;
    double* d_yPosition;
    double* d_inPressure;
    double* d_inVolume;
    double* d_inSoundSpeed;
    double* d_inVelocity;
    int* d_neighbourlist0;
    int* d_neighbourlist1;
    int* d_neighboursize0;
    int* d_neighboursize1;
    int* d_LPFOrder0;
    int* d_LPFOrder1;
   
// device arrays which dont need memcopy
    int* d_numRow;
    int* d_numCol;
   
   
   
   
    hipMalloc((void**)&d_xPosition,sizeof(double)*numParticle);
    hipMalloc((void**)&d_yPosition,sizeof(double)*numParticle);
    hipMalloc((void**)&d_inPressure,sizeof(double)*numParticle);
    hipMalloc((void**)&d_inVolume,sizeof(double)*numParticle );
    hipMalloc((void**)&d_inVelocity, sizeof(double)*numParticle);
    hipMalloc((void**)&d_inSoundSpeed,sizeof(double)*numParticle);
    hipMalloc((void**)&d_neighbourlist0,sizeof(int)*numFluid*numNeighbourone);
    hipMalloc((void**)&d_neighbourlist1,sizeof(int)*numFluid*numNeighbourone);
    hipMalloc((void**)&d_neighboursize0,sizeof(int)*numFluid);
    hipMalloc((void**)&d_neighboursize1,sizeof(int)*numFluid);
    hipMalloc((void**)&d_LPFOrder0,sizeof(int)*numFluid);
    hipMalloc((void**)&d_LPFOrder1,sizeof(int)*numFluid);
    hipMalloc((void**)&d_numRow,sizeof(int)*numFluid);
    hipMalloc((void**)&d_numCol,sizeof(int)*numFluid);
cout<<"-------------------------cuda allocate done----------------------------------"<<endl;

//memory copy

    hipMemcpy(d_xPosition,xPosition,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_yPosition,yPosition,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inPressure,inPressure,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inVolume,inVolume,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inVelocity,inVelocity,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inSoundSpeed,inSoundSpeed,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist0,neighbourlist0,sizeof(int)*numFluid*numNeighbourone,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist1,neighbourlist1,sizeof(int)*numFluid*numNeighbourone,hipMemcpyHostToDevice);
    hipMemcpy(d_neighboursize0,neighboursize0,sizeof(int)*numFluid,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist1,neighboursize1,sizeof(int)*numFluid,hipMemcpyHostToDevice);
    hipMemcpy(d_LPFOrder0,LPFOrder0,sizeof(int)*numFluid,hipMemcpyHostToDevice);
    hipMemcpy(d_LPFOrder1,LPFOrder1,sizeof(int)*numFluid,hipMemcpyHostToDevice);



    
    
   cout<<"----------------------------mem allocate and copy done--------------------------------"<<endl; 
    
    
    //----------------OUTPUT-------------------------
/*    
    double* outVelocity = new double[numParticle];
    double* outPressure = new double[numParticle];
    double* outSoundSpeed = new double[numParticle];
    double* outVolume = new double[numParticle];

*/



  cout<<"--------------------------------Testing---------------------------------------"<<endl;  

    dim3 blocks(128,1);
    dim3 threads(128,1);
    computeRowandCol<<<blocks,threads>>>(d_neighboursize0,d_numRow,d_numCol,d_LPFOrder0,numFluid);


//build double device pointer A
    double** A;
    hipMalloc((void**)&A,sizeof(double*)*numFluid);
    double** A_temp = new double*[numFluid];
   for(int i=0;i<numFluid;i++){
        hipMalloc((void**)&A_temp[i],sizeof(double)*5*numNeighbourone);
    }
    hipMemcpy(A, A_temp,sizeof(double*)*numFluid,hipMemcpyHostToDevice);
//build distance array

    double* d_distance;
    hipMalloc((void**)&d_distance,sizeof(double)*numFluid);
  
    cout<<"------------------------------Testing2---------------------------"<<endl;
    dim3 blocks1(128,1);
    dim3 threads1(128,1);
 


 
  
    computeA2D<<<blocks1,threads1>>>(d_neighbourlist0,d_LPFOrder0,d_numRow,d_xPosition,d_yPosition, numFluid,numNeighbourone,A,d_distance);

    
    cout<<"-----------------------Testing Done------------------------------"<<endl;
/*    
for(int i=0;i<numFluid;i++){
   cout<<"A of number: "<<i<<endl;
   magma_dprint_gpu(6,1,A_temp[i],6,queue_qr);
   }
*/
    

    

//Process QR batched mode
   
   
   
    magma_int_t m = 3;
    magma_int_t n = 2;
    magma_int_t lda = 3;
    magma_int_t min_mn = min(m,n);
    double **Tau;
    hipMalloc((void**)&Tau,numFluid*sizeof(double*));
    double** Tau_temp = new double*[numFluid];
    for(int i=0;i<numFluid;i++){
        hipMalloc((void**)&Tau_temp[i],sizeof(double)*min_mn);
    }



    hipMemcpy(Tau, Tau_temp, sizeof(double*)*numFluid, hipMemcpyHostToDevice);  
    magma_int_t* info;

    hipMalloc((void**)&info,numFluid*sizeof(magma_int_t));
    
    magma_int_t batchid = numFluid;

    //Start QR
  
    magma_dgeqrf_batched(m,n,A,lda,Tau,info,batchid,queue_qr);
  

  
  
    cout<<"-------------------------QR DONE----------------------------------"<<endl;





    //build right hand side B
    double** B;
    hipMalloc((void**)&B,sizeof(double*)*numFluid);
    double** B_temp = new double*[numFluid];
    for(int i=0;i<numFluid;i++){
        hipMalloc((void**)&B_temp[i],sizeof(double)*numNeighbourone);
    }
    hipMemcpy(B,B_temp,sizeof(double*)*numFluid,hipMemcpyHostToDevice);
    computeB<<<blocks,threads>>>(d_neighbourlist0, d_numRow, d_inPressure, numNeighbourone, numFluid, B);
    /*
for(int i=0;i<numFluid;i++){
   cout<<"number: "<<i<<endl;
   magma_dprint_gpu(3,1,B_temp[i],3,queue_qr);
   }
*/


//solver linear system

    double **result;
    hipMalloc((void**)&result,numFluid*sizeof(double*));
    double** result_temp = new double*[numFluid];
    for(int i=0;i<numFluid;i++){
        hipMalloc((void**)&result_temp[i],sizeof(double)*min_mn);
    }



    hipMemcpy(result, result_temp, sizeof(double*)*numFluid, hipMemcpyHostToDevice);  
   
    computeLS<<<blocks,threads>>>(A,B,Tau,d_numRow,d_numCol,numFluid, result);

cout<<"BIG SUCCESS!!"<<endl;
for(int i=0;i<numFluid;i++){
   cout<<"number: "<<i<<endl;
   magma_dprint_gpu(2,1,result_temp[i],2,queue_qr);
}   
    magma_queue_destroy(queue_qr);
    magma_finalize();
//release memory


    delete[] inPressure;
    delete[] inVolume;
    delete[] inVelocity;
    delete[] inSoundSpeed;
    delete[] neighbourlist0;
    delete[] neighbourlist1;
    delete[] neighboursize0;
    delete[] neighboursize1;
    delete[] LPFOrder0;
    delete[] LPFOrder1;
    delete[] A_temp;
    delete[] xPosition;
    delete[] yPosition;
    delete[] B_temp;
    delete[] result_temp;
    
    hipFree(d_neighboursize0);
    hipFree(d_neighboursize1);
    hipFree(d_neighbourlist0);
    hipFree(d_neighbourlist1);
    hipFree(d_LPFOrder0);
    hipFree(d_LPFOrder1);
    hipFree(d_inPressure);
    hipFree(d_inVolume);
    hipFree(d_inSoundSpeed);
    hipFree(d_inVelocity);
    hipFree(d_numRow);
    hipFree(d_numCol);
    hipFree(A);
    hipFree(result);
    for(int i=0;i<numParticle;i++){
        hipFree(A_temp[i]);
        hipFree(B_temp[i]);
        hipFree(result_temp[i]);
    }
    hipFree(d_distance);
    hipFree(B);
    

// QR
    delete[] Tau_temp;
    hipFree(Tau);
    for(int i=0;i<numParticle;i++){
        hipFree(Tau_temp[i]);
    }
    hipFree(info);
}

    

