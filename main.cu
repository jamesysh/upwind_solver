#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include "matrix_build.h"
#include <hipblas.h>
#include <magma_lapack.h>
#include <magma_v2.h>
using namespace std;



int main() {
    
    
    magma_init();


    int numParticle = 2268;
    int numNeighbourone = 177;
    int numNeighbour = 177;
// read data from txt file
    double* inPressure = new double[numParticle];
    double* inVolume = new double[numParticle];
    double* inSoundSpeed = new double[numParticle];
    double* inVelocity = new double[numParticle];
    int* neighbourlist0 = new int[numParticle*numNeighbourone];
    int* neighbourlist1 = new int[numParticle*numNeighbourone];
    int* neighboursize0 =new int[numParticle];
    int* neighboursize1 = new int[numParticle];
    int* LPFOrder0 = new int[numParticle];
    int* LPFOrder1 = new int[numParticle];
    double* xPosition = new double[numParticle];
    double* yPosition = new double[numParticle];
    
    
    //store data into array

    ifstream myfile;
    
    myfile.open("xPosition.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        xPosition[i] = tem;
    }
    myfile.close();

    myfile.open("yPosition.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        yPosition[i] = tem;
    }
    myfile.close();


   myfile.open("inPressure.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        inPressure[i]=tem;
    }
    myfile.close();
    
    myfile.open("inVelocity.txt");
    for(int i=0;i<numParticle;i++){
        double tem;
        myfile>>tem;
        inVelocity[i]=tem;
    }
    myfile.close();
    
    myfile.open("inSoundSpeed.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        inSoundSpeed[i]=tem;
    }
    myfile.close();
   
    myfile.open("inVolume.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        inVolume[i]=tem;
    }
    myfile.close();
   
  myfile.open("neighbourlist0.txt");
    for(int i=0;i<numParticle*numNeighbourone;i++){
        int tem;
        myfile>>tem;
        neighbourlist0[i]=tem;
    }
    myfile.close();
    myfile.open("neighbourlist1.txt");
    for(int i=0;i<numParticle*numNeighbourone;i++){
        int tem;
        myfile>>tem;
        neighbourlist1[i]=tem;
    }
    myfile.close();
      myfile.open("neighboursize0.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        neighboursize0[i]=tem;
    }
    myfile.close();
     myfile.open("neighboursize1.txt");
    for(int i=0;i<numParticle;i++){
       double tem;
        myfile>>tem;
        neighboursize1[i]=tem;
    }
    myfile.close();


    fill_n(LPFOrder0,numParticle,1);
    fill_n(LPFOrder1,numParticle,1);

//device arrays which need copy
    
    double* d_xPosition;
    double* d_yPosition;
    double* d_inPressure;
    double* d_inVolume;
    double* d_inSoundSpeed;
    double* d_inVelocity;
    int* d_neighbourlist0;
    int* d_neighbourlist1;
    int* d_neighboursize0;
    int* d_neighboursize1;
    int* d_LPFOrder0;
    int* d_LPFOrder1;
   
// device arrays which dont need memcopy
    int* d_numRow;
    int* d_numCol;
   
   
   
   
    hipMalloc((void**)&d_xPosition,sizeof(double)*numParticle);
    hipMalloc((void**)&d_yPosition,sizeof(double)*numParticle);
    hipMalloc((void**)&d_inPressure,sizeof(double)*numParticle);
    hipMalloc((void**)&d_inVolume,sizeof(double)*numParticle );
    hipMalloc((void**)&d_inVelocity, sizeof(double)*numParticle);
    hipMalloc((void**)&d_inSoundSpeed,sizeof(double)*numParticle);
    hipMalloc((void**)&d_neighbourlist0,sizeof(int)*numParticle*numNeighbourone);
    hipMalloc((void**)&d_neighbourlist1,sizeof(int)*numParticle*numNeighbourone);
    hipMalloc((void**)&d_neighboursize0,sizeof(int)*numParticle);
    hipMalloc((void**)&d_neighboursize1,sizeof(int)*numParticle);
    hipMalloc((void**)&d_LPFOrder0,sizeof(int)*numParticle);
    hipMalloc((void**)&d_LPFOrder1,sizeof(int)*numParticle);
    hipMalloc((void**)&d_numRow,sizeof(int)*numParticle);
    hipMalloc((void**)&d_numCol,sizeof(int)*numParticle);
cout<<"-------------------------cuda allocate done----------------------------------"<<endl;

//memory copy

    hipMemcpy(d_xPosition,xPosition,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_yPosition,yPosition,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inPressure,inPressure,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inVolume,inVolume,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inVelocity,inVelocity,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_inSoundSpeed,inSoundSpeed,sizeof(double)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist0,neighbourlist0,sizeof(int)*numParticle*numNeighbourone,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist1,neighbourlist1,sizeof(int)*numParticle*numNeighbourone,hipMemcpyHostToDevice);
    hipMemcpy(d_neighboursize0,neighboursize0,sizeof(int)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_neighbourlist1,neighboursize1,sizeof(int)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_LPFOrder0,LPFOrder0,sizeof(int)*numParticle,hipMemcpyHostToDevice);
    hipMemcpy(d_LPFOrder1,LPFOrder1,sizeof(int)*numParticle,hipMemcpyHostToDevice);



    
    
   cout<<"----------------------------mem allocate and copy done--------------------------------"<<endl; 
    
    
    //----------------OUTPUT-------------------------
/*    
    double* outVelocity = new double[numParticle];
    double* outPressure = new double[numParticle];
    double* outSoundSpeed = new double[numParticle];
    double* outVolume = new double[numParticle];

*/



  cout<<"--------------------------------Testing---------------------------------------"<<endl;  

    dim3 blocks(128,1);
    dim3 threads(128,1);
    computeRowandCol<<<blocks,threads>>>(d_neighboursize0,d_numRow,d_numCol,d_LPFOrder0,numParticle);


//build double device pointer A
    double** A;
    hipMalloc((void**)&A,sizeof(double*)*numParticle);
    double** A_temp = new double*[numParticle];
   for(int i=0;i<numParticle;i++){
        hipMalloc((void**)&A_temp[i],sizeof(double)*5*numNeighbourone);
    }
    hipMemcpy(A, A_temp,sizeof(double*)*numParticle,hipMemcpyHostToDevice);
//build distance array

    double* d_distance;
    hipMalloc((void**)&d_distance,sizeof(double)*numParticle);
  
    cout<<"------------------------------Testing2---------------------------"<<endl;
    dim3 blocks1(128,1);
    dim3 threads1(128,1);
 


 
  
    computeA2D<<<blocks1,threads1>>>(d_neighbourlist0,d_LPFOrder0,d_numRow,d_xPosition,d_yPosition, numParticle,numNeighbourone,A,d_distance);
    cout<<"-----------------------Testing Done------------------------------"<<endl;
/*    
    double* temp1 = new double[10];
    
    hipMemcpy(temp1,A_temp[2268],sizeof(double)*10,hipMemcpyDeviceToHost);
    for(int i=0;i<10;i++){
        cout<<temp1[i]<<endl;
    }

 */

//Process QR batched mode

    magma_int_t dev_t = 0;
    magma_queue_t queue_qr = NULL;
    magma_queue_create(dev_t,&queue_qr);
   
   //set up device and queue
   
   
   
    magma_int_t m = 3;
    magma_int_t n = 2;
    magma_int_t lda = 3;
    magma_int_t min_mn = min(m,n);
    double **Tau;
    hipMalloc((void**)&Tau,numParticle*sizeof(double*));
    double** Tau_temp = new double*[numParticle];
    for(int i=0;i<numParticle;i++){
        hipMalloc((void**)&Tau_temp[i],sizeof(double)*min_mn);
    }



    hipMemcpy(Tau, Tau_temp, sizeof(double*)*numParticle, hipMemcpyHostToDevice);  
    magma_int_t* info;

    hipMalloc((void**)&info,numParticle*sizeof(magma_int_t));
    
    magma_int_t batchid = numParticle;

    //Start QR
  
    magma_dgeqrf_batched(m,n,A,lda,Tau,info,batchid,queue_qr);
  

  
  
    cout<<"-------------------------QR DONE----------------------------------"<<endl;

//release memory


    delete[] inPressure;
    delete[] inVolume;
    delete[] inVelocity;
    delete[] inSoundSpeed;
    delete[] neighbourlist0;
    delete[] neighbourlist1;
    delete[] neighboursize0;
    delete[] neighboursize1;
    delete[] LPFOrder0;
    delete[] LPFOrder1;
    delete[] A_temp;
    delete[] xPosition;
    delete[] yPosition;
    

    
    hipFree(d_neighboursize0);
    hipFree(d_neighboursize1);
    hipFree(d_neighbourlist0);
    hipFree(d_neighbourlist1);
    hipFree(d_LPFOrder0);
    hipFree(d_LPFOrder1);
    hipFree(d_inPressure);
    hipFree(d_inVolume);
    hipFree(d_inSoundSpeed);
    hipFree(d_inVelocity);
    hipFree(d_numRow);
    hipFree(d_numCol);
    hipFree(A);
    for(int i=0;i<numParticle;i++){
        hipFree(A_temp[i]);
    }
    hipFree(d_distance);
    

// QR
    delete[] Tau_temp;
    hipFree(Tau);
    for(int i=0;i<numParticle;i++){
        hipFree(Tau_temp[i]);
    }
    hipFree(info);
}



