#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix_build.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void computeRowandCol(const int*neighboursize, int*numRow, int*numCol,int* LPFOrder,const int numParticle)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   int offset = blockDim.x*gridDim.x;
   int numrow2nd = 36;
   int numrow1st = 3;
   int numcol2nd = 5;
   int numcol1st = 2;
   while(tid<numParticle){
    
   int numNeisize =  neighboursize[tid];
       if(LPFOrder[tid]==2){
           if(numNeisize >= numrow2nd){
               numRow[tid] = numrow2nd;
               numCol[tid] = numcol2nd;
           }
            else LPFOrder[tid] = 1;
       }
    
       if(LPFOrder[tid]==1){
           if(numNeisize >= numrow1st){
               numRow[tid] = numrow1st;
               numCol[tid] = numcol1st;
           }
            else LPFOrder[tid] = 0;


       }
        
       if(LPFOrder[tid]==0){
            numRow[tid] = 0;
            numCol[tid] = 0;

       }
        tid = tid + offset;
}

}



__global__ void computeA2D(const int*neighbourList,const int*LPFOrder,const int* numRow,const double*x,const double*y, const int numParticle,const int maxNeighbourOneDir,double**A,double*dis)
    {
        
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x*gridDim.x;
    //printf("runing form %d\n",tid);
    while(tid<numParticle){
        int numOfRow = numRow[tid];
        double distance = sqrt((x[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-x[tid])*(x[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-x[tid])+(y[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-y[tid])*(y[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-y[tid]));
        if(LPFOrder[tid] == 1){
            
            for(int i=0;i<numOfRow;i++){
            
                int neiIndex = neighbourList[tid*maxNeighbourOneDir+i];
                    
                double h = (x[neiIndex]-x[tid])/distance;
                
                double k = (y[neiIndex]-y[tid])/distance;
                A[tid][i] = h;
                A[tid][i+numOfRow] = k;
            }   
    
        }
        else if(LPFOrder[tid] == 2){
            for(int i=0;i<numOfRow;i++){
                int neiIndex = neighbourList[tid*maxNeighbourOneDir+i];
                double h = (x[neiIndex]-x[tid])/distance;
                double k = (y[neiIndex]-y[tid])/distance;
                A[tid][i] = h;
                A[tid][i + numOfRow] = k;
                A[tid][i + 2*numOfRow] = 0.5*h*h;
                A[tid][i + 3*numOfRow] = 0.5*k*k;
                A[tid][i + 4*numOfRow] = h*k;


            }
        
        } 
    dis[tid] = distance;
    tid = tid + offset;
    }
}

__global__ void computeB(const int* neighbourList, const int* numRow, const double* inData, const int maxNumNeighbourOne, const int numParticle,
                        double** b)//output vector b

{
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x*gridDim.x;
    while(tid<numParticle){
        for(int i=0;i<numRow[tid];i++){
            int neiIndex = neighbourList[tid*maxNumNeighbourOne + i];
            b[tid][i] = inData[neiIndex] - inData[tid];
        }
    
        tid = tid + offset;
    }
}


__global__ void computeLS(double**A,double**B,double**Tau, const int* numRow,const int* numCol ,const int numFluid, 
                        double**Result)//output result
{

    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x*gridDim.x;
    while(tid < numFluid){
        int nrow = numRow[tid];
        int ncol = numCol[tid];
        for(int i=0;i<ncol;i++){
//cant build v here cauz we need a fixed size in cuda kernal    no double v[ncol]
            double v_times_b = 0.;
            for(int j=0;j<nrow;j++){
                if(j < i) continue;
                if(j == i) v_times_b += 1*B[tid][j];
                else v_times_b += A[tid][j+i*nrow]*B[tid][j];
            }
            v_times_b *= Tau[tid][i];

            for(int j=0;j<nrow;j++){
                if(j < i) continue;
                if(j == i) B[tid][j] -= v_times_b;
                else
                B[tid][j] -= v_times_b*A[tid][j+i*nrow];
            }

        }

//compute QTB complete

//Backsubstitution
        for(int i=ncol-1;i>=0;i--){
            Result[tid][i] = B[tid][i]/A[tid][i*nrow+i];
            for(int j=0;j<i;j++){
                
                B[tid][j] -= A[tid][j+i*nrow]*Result[tid][i];
            }


        }
    tid += offset;

    }
}























