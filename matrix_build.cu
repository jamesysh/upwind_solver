#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix_build.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void computeRowandCol(const int*neighboursize, int*numRow, int*numCol,int* LPFOrder,const int numParticle){
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   int offset = blockDim.x*gridDim.x;
   int numrow2nd = 36;
   int numrow1st = 3;
   int numcol2nd = 5;
   int numcol1st = 2;
   while(tid<numParticle){
    
   int numNeisize =  neighboursize[tid];
       if(LPFOrder[tid]==2){
           if(numNeisize >= numrow2nd){
               numRow[tid] = numrow2nd;
               numCol[tid] = numcol2nd;
           }
            else LPFOrder[tid] = 1;
       }
    
       if(LPFOrder[tid]==1){
           if(numNeisize >= numrow1st){
               numRow[tid] = numrow1st;
               numCol[tid] = numcol1st;
           }
            else LPFOrder[tid] = 0;


       }
        
       if(LPFOrder[tid]==0){
            numRow[tid] = 0;
            numCol[tid] = 0;

       }
        tid = tid + offset;
}

    __syncthreads();
}



__global__ void computeA2D(const int*neighbourList,const int*LPFOrder,const int* numRow,const double*x,const double*y, const int numParticle,const int maxNeighbourOneDir,double**A,double*dis)
    {
        
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x*gridDim.x;
    //printf("runing form %d\n",tid);
    while(tid<numParticle){
        int numOfRow = numRow[tid];
        double distance = sqrt((x[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-x[tid])*(x[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-x[tid])+(y[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-y[tid])*(y[neighbourList[tid*maxNeighbourOneDir+numOfRow/4]]-y[tid]));
        if(LPFOrder[tid] == 1){
            
            for(int i=0;i<numOfRow;i++){
            
                int neiIndex = neighbourList[tid*maxNeighbourOneDir+i];
                    
                double h = (x[neiIndex]-x[tid])/distance;
                
                double k = (y[neiIndex]-y[tid])/distance;
                A[tid][i] = h;
                A[tid][i+numOfRow] = k;
            }   
    
        }
        else if(LPFOrder[tid] == 2){
            for(int i=0;i<numOfRow;i++){
                int neiIndex = neighbourList[tid*maxNeighbourOneDir+i];
                double h = (x[neiIndex]-x[tid])/distance;
                double k = (y[neiIndex]-y[tid])/distance;
                A[tid][i] = h;
                A[tid][i + numOfRow] = k;
                A[tid][i + 2*numOfRow] = 0.5*h*h;
                A[tid][i + 3*numOfRow] = 0.5*k*k;
                A[tid][i + 4*numOfRow] = h*k;


            }
        
        } 
    dis[tid] = distance;
    tid = tid + offset;
    }
}



























